#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        


        // TODO: __global__

        __global__ void kernNaiveScan(int n, int* odata, const int* idata, int stride)
        {
            int index = threadIdx.x + blockDim.x * blockIdx.x;
            if (index < n)
            {

                if (index >= (1 << stride))
                {
                    odata[index] = idata[index] + idata[index - (1 << stride)];
                }
                else
                {
                    odata[index] = idata[index];
                }
            }
        }

        __global__ void kernInclusiveToExclusive(int n, int* odata, const int* idata)
        {
            int index = threadIdx.x + blockDim.x * blockIdx.x;
            if (index == 0)
            {
                odata[0] = 0;
            }
            else if (index < n)
            {
                odata[index] = idata[index - 1];
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            // TODO
            //Starting Input
            int *dev_arrA;
            //Starting Output
            int *dev_arrB;

            hipMalloc((void**)&dev_arrA, sizeof(int) * n);
            hipMalloc((void**)&dev_arrB, sizeof(int) * n);

            hipMemcpy(dev_arrA, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            

            int threadsPerBlock = 256;
            dim3 totalBlocks ((n + threadsPerBlock - 1) / threadsPerBlock);

            int log2Ceil = ilog2ceil(n);

            timer().startGpuTimer();
            for (int i = 0; i < log2Ceil; i++)
            {
                kernNaiveScan << <totalBlocks, threadsPerBlock >> > (n, dev_arrB, dev_arrA, i);
                std::swap(dev_arrA, dev_arrB);
            }
            
            kernInclusiveToExclusive <<<totalBlocks, threadsPerBlock >>> (n, dev_arrB, dev_arrA);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_arrB, sizeof(int) * n, hipMemcpyDeviceToHost);
            


            hipFree(dev_arrA);
            hipFree(dev_arrB);

            
        }
    }
}
